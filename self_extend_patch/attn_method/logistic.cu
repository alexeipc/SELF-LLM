#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

const int MAX_THREAD = 1000;

__device__ double gs_inverse_generating_function(int y, double rate, double capacity) {
    double numerator = std::log(y * capacity - y) - std::log(capacity - y);
    double denominator = rate;
    return (double) numerator / denominator;
}

struct Group {
    int first;
    int last;

    __device__ Group(int first, int last)
        : first(first), last(last) // Use initializer list
    {
    }
};

__global__ void gpu_key_group_id(int n, int capacity, int presum, int last_group_size, Group* groups, int* res) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int id = presum + tid;

    if (id >= n) return;

    int next_group_size = last_group_size + 1;
    int last_group_pos = groups[last_group_size].last;
    int group_id_val = last_group_pos + (id - presum + next_group_size - 1) / next_group_size;

    int max_n = presum + next_group_size * (groups[last_group_size + 1].last - groups[last_group_size + 1].first + 1);
    if (id > max_n && next_group_size != capacity - 1) return;

    if (n - id >= 0 && n >= group_id_val && n - id < n) {
		//printf("id=%d, group_id=%d, n-id=%d\n", id, group_id_val, n - id);
		//printf("%p\n", res);
		//printf("%d\n", res[0]);
	    res[id] = group_id_val;

    }
}

__global__ void gpu_query_group_id(int n, int window_size, int* group_query_position, int* group_key_position) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;

    if (i < window_size) group_query_position[i] = window_size;
    else {
        group_query_position[i] = window_size + group_key_position[i - window_size];
    }
}

__global__ void freq_group(int capacity, double rate, Group* groups) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= 1 && i < capacity - 1) {
		double lower_bound = gs_inverse_generating_function(i, rate, capacity);
        double upper_bound = gs_inverse_generating_function(i + 1, rate, capacity);

		groups[i] = Group(ceil(lower_bound), floor(upper_bound));

		if (upper_bound == (double)floor(upper_bound)) groups[i].last--;
   }
}

void async_generator(torch::Tensor group_query_position, torch::Tensor group_key_position, int n, int window_size, double rate, double capacity) {
	Group* groups;
	int tensor_device = group_key_position.device().index();
hipSetDevice(tensor_device);
	hipMallocManaged(&groups, (capacity + 1) * sizeof(Group));
	hipMemset(groups, 0, (capacity + 1) * sizeof(Group));

	freq_group<<<1, capacity - 1>>>(capacity, rate, groups);
	hipDeviceSynchronize();


groups[0].last = -1;

    int presum = 0;
	for (int i = 1; i < capacity; ++i) {
        int group_size = groups[i].last - groups[i].first + 1;
        int next_group_size = groups[i + 1].last - groups[i + 1].first + 1;

	if (group_size <= 0 || next_group_size <= 0) continue;
	gpu_key_group_id<<<(n + MAX_THREAD - 1)/MAX_THREAD, MAX_THREAD>>>(n, capacity, presum, i - 1, groups ,group_key_position.data_ptr<int>());
	presum = presum + i * group_size;
    }
	hipDeviceSynchronize();

    gpu_query_group_id<<<(n + MAX_THREAD - 1)/MAX_THREAD, MAX_THREAD>>>(n, window_size, group_query_position.data_ptr<int>() , group_key_position.data_ptr<int>());
    hipDeviceSynchronize();
    hipFree(groups);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("async_generator", &async_generator, "Description of your function");
}
